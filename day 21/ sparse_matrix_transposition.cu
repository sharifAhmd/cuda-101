
#include <hip/hip_runtime.h>
#include <iostream>

#define NNZ 6  
/*
It transposes a sparse matrix in Coordinate List(COO) format by swapping row and column indices of 
all non-zero entries. Each thread processes one element, demonstrating indirect memory access and 
gather-like behavior.
*/
__global__ void transposeCOO(int *row_idx, int *col_idx, float *values,
                             int *t_row_idx, int *t_col_idx, float *t_values) {
    int i = threadIdx.x;

    t_row_idx[i] = col_idx[i];
    t_col_idx[i] = row_idx[i];
    t_values[i]   = values[i];
}

int main() {

    int h_row_idx[NNZ] = {0, 0, 2, 2, 2, 0};  
    int h_col_idx[NNZ] = {0, 2, 0, 1, 1, 1}; 
    float h_values[NNZ] = {5, 8, 3, 6, 1, 4}; 

    int h_t_row_idx[NNZ], h_t_col_idx[NNZ];
    float h_t_values[NNZ];

    int *d_row_idx, *d_col_idx, *d_t_row_idx, *d_t_col_idx;
    float *d_values, *d_t_values;

    hipMalloc(&d_row_idx, NNZ * sizeof(int));
    hipMalloc(&d_col_idx, NNZ * sizeof(int));
    hipMalloc(&d_values, NNZ * sizeof(float));
    hipMalloc(&d_t_row_idx, NNZ * sizeof(int));
    hipMalloc(&d_t_col_idx, NNZ * sizeof(int));
    hipMalloc(&d_t_values, NNZ * sizeof(float));

    hipMemcpy(d_row_idx, h_row_idx, NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_col_idx, NNZ * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, NNZ * sizeof(float), hipMemcpyHostToDevice);

    transposeCOO<<<1, NNZ>>>(d_row_idx, d_col_idx, d_values,
                              d_t_row_idx, d_t_col_idx, d_t_values);

    hipMemcpy(h_t_row_idx, d_t_row_idx, NNZ * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_t_col_idx, d_t_col_idx, NNZ * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_t_values, d_t_values, NNZ * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Transposed COO Entries:\n";
    for (int i = 0; i < NNZ; i++) {
        std::cout << "Row: " << h_t_row_idx[i]
                  << ", Col: " << h_t_col_idx[i]
                  << ", Val: " << h_t_values[i] << "\n";
    }

    hipFree(d_row_idx);
    hipFree(d_col_idx);
    hipFree(d_values);
    hipFree(d_t_row_idx);
    hipFree(d_t_col_idx);
    hipFree(d_t_values);

    return 0;
}
