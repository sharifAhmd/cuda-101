
#include <hip/hip_runtime.h>
#include <iostream>
#define WIDTH 8
#define HEIGHT 8

/**
 * Performs a 2D convolution operation on the input array.
 */
__global__ void stencil2D(int *input, int *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int idx = y * width + x;
        output[idx] = input[idx] + 
                      input[(y - 1) * width + x] + // top
                      input[(y + 1) * width + x] + // bottom
                      input[y * width + (x - 1)] + // left
                      input[y * width + (x + 1)];  // right
    }
}

int main() {
    int size = WIDTH * HEIGHT;
    int h_input[size], h_output[size];

    for (int i = 0; i < size; i++) {
        h_input[i] = 1;  
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(int));

    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(8, 8);
    dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    stencil2D<<<numBlocks, threadsPerBlock>>>(d_input, d_output, WIDTH, HEIGHT);

    hipMemcpy(h_output, d_output, size * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Stencil output:\n";
    for (int y = 0; y < HEIGHT; y++) {
        for (int x = 0; x < WIDTH; x++) {
            std::cout << h_output[y * WIDTH + x] << " ";
        }
        std::cout << "\n";
    }

    hipFree(d_input);
    hipFree(d_output);
    return 0;
}
