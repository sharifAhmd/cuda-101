#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int n, float alpha) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        output[idx] = (x > 0.0f) ? x : alpha * x;
    }
}

void print_array(const char* label, const float* arr, int n) {
    printf("%s: [", label);
    for (int i = 0; i < n; ++i)
        printf(" %.6f", arr[i]);
    printf(" ]\n");
}

int main() {
    const int N = 16;
    float alpha = 0.01f;
    float h_in[N], h_out[N];
    for (int i = 0; i < N; ++i)
        h_in[i] = i - 8;

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 128, blocks = (N + threads - 1) / threads;
    leaky_relu_kernel<<<blocks, threads>>>(d_in, d_out, N, alpha);
    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    print_array("Input", h_in, N);
    print_array("LeakyReLU", h_out, N);

    hipFree(d_in); hipFree(d_out);
    return 0;
}
