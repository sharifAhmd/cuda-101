#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define M 128  
#define K 64   
#define N 32  

__global__ void dense_layer(const float* X, const float* W, float* Y, int M, int K, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += X[row * K + k] * W[col * K + k];  
        }
        Y[row * N + col] = sum;
    }
}

int main() {
    float *h_X = new float[M * K];
    float *h_W = new float[N * K];
    float *h_Y = new float[M * N];

    // Initialize X and W with simple data
    for (int i = 0; i < M * K; ++i) h_X[i] = 1.0f;
    for (int i = 0; i < N * K; ++i) h_W[i] = 0.5f;

    float *d_X, *d_W, *d_Y;
    hipMalloc(&d_X, M * K * sizeof(float));
    hipMalloc(&d_W, N * K * sizeof(float));
    hipMalloc(&d_Y, M * N * sizeof(float));

    hipMemcpy(d_X, h_X, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, N * K * sizeof(float), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (M + threads.y - 1) / threads.y);
    dense_layer<<<blocks, threads>>>(d_X, d_W, d_Y, M, K, N);
    hipDeviceSynchronize();

    hipMemcpy(h_Y, d_Y, M * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "First 5 output values:\n";
    for (int i = 0; i < 5; ++i) {
        std::cout << h_Y[i] << " ";
    }
    std::cout << "\n";

    hipFree(d_X);
    hipFree(d_W);
    hipFree(d_Y);
    delete[] h_X;
    delete[] h_W;
    delete[] h_Y;

    return 0;
}

