
#include <hip/hip_runtime.h>
#include <iostream>
#include <climits>
#define N 5
#define E 8
#define INF 1e9

__global__ void relax_edges(int *src, int *dst, int *weights, int *dist, bool *updated) {
    int tid = threadIdx.x;
    int u = src[tid];
    int v = dst[tid];
    int w = weights[tid];

    if (dist[u] != INF && dist[u] + w < dist[v]) {
        dist[v] = dist[u] + w;
        *updated = true;
    }
}

int main() {
    int h_src[E]    = {0, 0, 1, 1, 1, 2, 3, 4};
    int h_dst[E]    = {1, 2, 2, 3, 4, 4, 4, 3};
    int h_weights[E]= {6, 7, 5, -4, 8, -3, 9, 7};

    int h_dist[N];
    for (int i = 0; i < N; i++) h_dist[i] = INF;
    h_dist[0] = 0; 

    int *d_src, *d_dst, *d_weights, *d_dist;
    bool h_updated, *d_updated;

    hipMalloc(&d_src, E * sizeof(int));
    hipMalloc(&d_dst, E * sizeof(int));
    hipMalloc(&d_weights, E * sizeof(int));
    hipMalloc(&d_dist, N * sizeof(int));
    hipMalloc(&d_updated, sizeof(bool));

    hipMemcpy(d_src, h_src, E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dst, h_dst, E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, h_weights, E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dist, h_dist, N * sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < N - 1; i++) {
        h_updated = false;
        hipMemcpy(d_updated, &h_updated, sizeof(bool), hipMemcpyHostToDevice);

        relax_edges<<<1, E>>>(d_src, d_dst, d_weights, d_dist, d_updated);

        hipMemcpy(&h_updated, d_updated, sizeof(bool), hipMemcpyDeviceToHost);
        if (!h_updated) break; // Early exit if no updates
    }

    hipMemcpy(h_dist, d_dist, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Shortest distances from node 0:\n";
    for (int i = 0; i < N; i++) {
        std::cout << "Node " << i << ": ";
        if (h_dist[i] == INF) std::cout << "INF\n";
        else std::cout << h_dist[i] << "\n";
    }

    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_weights);
    hipFree(d_dist);
    hipFree(d_updated);
    return 0;
}
