#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void elu_kernel(const float* input, float* output, int n, float alpha) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        float x = input[i];
        output[i] = (x > 0.0f) ? x : alpha * (expf(x) - 1.0f);
    }
}

void print_array(const char* label, const float* arr, int n) {
    printf("%s: [", label);
    for (int i = 0; i < n; ++i)
        printf(" %.6f", arr[i]);
    printf(" ]\n");
}

int main() {
    const int N = 16;
    float alpha = 1.0f;
    float h_in[N], h_out[N];
    for (int i = 0; i < N; ++i)
        h_in[i] = i - 8; // Sample input: -8 to 7

    float *d_in, *d_out;
    hipMalloc(&d_in, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));
    hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

    int threads = 128, blocks = (N + threads - 1) / threads;
    elu_kernel<<<blocks, threads>>>(d_in, d_out, N, alpha);
    hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    print_array("Input", h_in, N);
    print_array("ELU", h_out, N);

    hipFree(d_in); hipFree(d_out);
    return 0;
}
