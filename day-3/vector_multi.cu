
#include <hip/hip_runtime.h>
#include <iostream>

#define N 8 


__global__ void vectorMultiply(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        c[idx] = a[idx] * b[idx];
    }
}


int main() {
    int a[N], b[N], c[N];        
    int *d_a, *d_b, *d_c;          


    for (int i = 0; i < N; i++) {
        a[i] = i + 1;              
        b[i] = (i + 1) * 2;        
    }


    hipMalloc((void**)&d_a, N * sizeof(int));
    hipMalloc((void**)&d_b, N * sizeof(int));
    hipMalloc((void**)&d_c, N * sizeof(int));


    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    vectorMultiply<<<2, 4>>>(d_a, d_b, d_c); 

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Vector Multiplication Results:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " * " << b[i] << " = " << c[i] << std::endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
